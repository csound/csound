#include "hip/hip_runtime.h"
// -*- c++ -*-
/*
  slindingm.cu:

  Copyright (C) 2014 Russell Bradford, Victor Lazzarini, John ffitch

  This file is part of Csound.

  The Csound Library is free software; you can redistribute it
  and/or modify it under the terms of the GNU Lesser General Public
  License as published by the Free Software Foundation; either
  version 2.1 of the License, or (at your option) any later version.

  Csound is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU Lesser General Public License for more details.

  You should have received a copy of the GNU Lesser General Public
  License along with Csound; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA
  02111-1307 USA
*/
// slindingm.cu
// experimental cuda opcodes
// using the sliding DFT
// adapted from Russell Bradford & John ffitch
//

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <inttypes.h>
#include <math.h>
#include <inttypes.h>
#include <sys/time.h>
#include <csdl.h>

//#define RB

#ifndef RB
typedef float real;
#else
typedef double real;
#endif

#define real(x) x
#define TWO_PI (2.0*M_PI)
#define NBATCH (256)

#define Re(z) ((z).x)
#define Im(z) ((z).y)
#define R(p) ((p).x)
#define Theta(p) ((p).y)

#define RectA (real(1.0))
#define RectB (real(0.0))
#define HannA (real(0.5))
#define HannB (real(-0.5))
#define A HannA
#define B2 (HannB/real(2.0))
#define WINDOW


typedef double2 complex;
typedef double2 phasor;

// Put a few things into constant memory to free up a couple of registers
//__constant__ int N, N2;
__constant__ real binbandwidth, nyquist;
// array alignment
__constant__ int offset;
// process in batches of this size 
__constant__ int nbatch;


#define CUDA(name, args ...) { name(args); checkcuda(csound, #name, __LINE__); }
#define CHK(name) checkcuda(csound, #name, __LINE__)
#define NEXT(p) p += offset

void checkcuda(CSOUND *csound, char* name, int line)
{
  hipError_t err = hipGetLastError();

  if (err) {
    csound->Message(csound, "%s: %s\nLine: %d\n", name, hipGetErrorString(err), line);
    exit(1);
  }
}

__device__ double atomicAdd(double* address, double val) { 
  unsigned long long int* address_as_ull = (unsigned long long int*)address; 
  unsigned long long int old = *address_as_ull, assumed; 
  do { assumed = old; 
    old = atomicCAS(address_as_ull, 
		    assumed, __double_as_longlong(val + __longlong_as_double(assumed))); 
  } while (assumed != old); 
  return __longlong_as_double(old); 
}

__device__ complex conjugate(complex z)
{
  return (complex){ Re(z), -Im(z) };
}

// round up to multiple of 256
__host__ __device__ int roundup(int n)
{
  return 1 + ((n - 1) | 0xff);
}

// Fin/Fout
// thread k does bin k; parallel across bins in a batch item..
__global__ void slide(real deltas[/*nbatch*/], complex Fin[/*N*/],
		      complex Fout[/*N*nbatch*/], 
                      real *sine, real *cosine, int N)
{
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  int b, N2 = N/2;
  complex f;
  real Fr, Fi, c, s;
  
  if (k > N2) return;
  f = Fin[k];

  // in constant memory
  s = sine[k];
  c = cosine[k];
  
  for (b = 0; b < nbatch; b++) {
    Fr = Re(f) + deltas[b];
    Fi = Im(f);
    f = (complex){ Fr*c - Fi*s, Fi*c + Fr*s };
    Fout[k] = f;
    NEXT(Fout);
  }
  Fin[k] = f;
}

__device__ real mod2Pi(real x)
{
  x = fmod(x, (real)TWO_PI);
  if (x > PI) return x - TWO_PI;
  if (x <= -PI) return x + TWO_PI;
  return x;
}

#ifdef RB
// below is one thread per bin, as per other kernels
// data is spread across blocks, but there is no sync across blocks
// thus cannot do in-place window
// parallel across bins in a batch item
// could do bins*batches
__global__ void window(complex Foutw[/*N*nbatch*/],
		       complex Fout[/*N*nbatch*/], int N)
{
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  int b, N2 = N/2;
  complex F, Fm1, Fp1, f;
  real Fr, Fi;

  // in the last block
  if (k > N2) return;

  for (b = 0; b < nbatch; b++) {
    F = Foutw[k];
    // conjugate reflection at edges
    Fp1 = k < N2 ? Foutw[k + 1] : conjugate(Foutw[N2-1]);
    Fm1 = k > 0 ? Foutw[k - 1] : conjugate(Foutw[1]);
    Fr = A*Re(F) + B2*(Re(Fm1) + Re(Fp1));
    Fi = A*Im(F) + B2*(Im(Fm1) + Im(Fp1));

    Re(f) = Fr;
    Im(f) = Fi;
    Fout[k] = f;

    NEXT(Fout);
    NEXT(Foutw);
  }
}

// parallel across batches
// <<<nbatch/256,256>>>
// what is memory access pattern?
// small array sum, so not worth a tree reduction?
// middle sample only, so +- on real parts
__global__ void reconstruct(complex f[/*nbatch*N*/], real s[/*nbatch*/], int N)
{
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  complex *F = f + offset*k;
  real sum = real(0.0);
  int bin, N2 = N/2;

  if (k >= nbatch) return;
  for (bin = 0; bin < N; bin += 2) {
    real v0 = bin <= N2 ? Re(F[bin]) : Re(F[N - bin]);
    real v1 = bin <  N2 ? Re(F[bin + 1]) : Re(F[N - bin - 1]);
    sum += v0 - v1;
  }
  s[k] = sum/N;
}

#else
// the code below depends on fast single-precision
// atomic addition

// this parallelises across N*nbatch
__global__ void window(complex Foutw[/*N*nbatch*/],
		       complex Fout[/*N*nbatch*/], int N)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  int N2 = N/2;
  int s = n/N;
  int k = n%N;
  complex F, Fm1, Fp1, f;
  real Fr, Fi;
  Foutw += s*offset;
  Fout += s*offset;

  // in the last block
  if (n > N*nbatch) return;

  F = Foutw[k];
  Fp1 = k < N2 ? Foutw[k + 1] : conjugate(Foutw[N2-1]);
  Fm1 = k > 0 ? Foutw[k - 1] : conjugate(Foutw[1]);

  Fr = A*Re(F) + B2*(Re(Fm1) + Re(Fp1));
  Fi = A*Im(F) + B2*(Im(Fm1) + Im(Fp1));

  Re(f) = Fr;
  Im(f) = Fi;
  Fout[k] = f;
}

// this parallelises across N2*nbatch
__global__ void reconstruct(complex f[/*nbatch*N*/], real s[/*nbatch*/], int N)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x, N2 = N/2;
  int k = n/N2;
  int bin = (n%N2)*2;
  complex *F = &f[offset*k];

  if (n >= nbatch*N2) return;

  real v0 = bin <= N2 ? Re(F[bin]) : Re(F[N - bin]);
  real v1 = bin <  N2 ? Re(F[bin + 1]) : Re(F[N - bin - 1]);
  atomicAdd(&s[k],v0-v1); 
}
#endif


__device__ phasor convert(complex f, real &oldiphase, int bin, int N)
{
  real Fr, Fi;
  real phase, deltaphase;
  phasor pha;

  Fr = Re(f);
  Fi = Im(f);

  // magnitude is easy
  R(pha) = hypot(Fi, Fr);

  phase = atan2(Fi, Fr);
  deltaphase = phase - oldiphase;
  oldiphase = phase;

  // subtract expected phase diff
  deltaphase = mod2Pi(deltaphase - bin*TWO_PI/N);

  // find actual freq
  // scale by bin width to give freq
  Theta(pha) = binbandwidth * (bin + deltaphase*N/TWO_PI);

  return pha;
}

__device__ phasor harmonic_shift(phasor pha, real fm)
{
  real sfreq;

  // tweak the freq using fm
  // same across all channels
  sfreq = Theta(pha)*fm;

  if (sfreq >= nyquist || sfreq <= -nyquist) R(pha) = real(0.0);
  Theta(pha) = sfreq;

  return pha;
}

__device__ real unconvert(phasor pha, real &oldophase, int N)
{
  real bin, phase;
  
  bin = Theta(pha)/binbandwidth;
  phase = bin*TWO_PI/N;
  phase = mod2Pi(oldophase + phase);
  oldophase = phase;

  return R(pha)*cos(phase);
}

// parallelism across bins*channels
__global__ void fmsyn(real inphase[/*N*/], complex F[/*nbatch*N*/],
		      real fm[/*nbatch*/], real outphase[/*N*/], int N)
{
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  int b, N2 = N/2;
  real oldiphase, oldophase, val;
  phasor pha;

  if (k > N2) return;

  oldiphase = inphase[k];
  oldophase = outphase[k];

  for (b = 0; b < nbatch; b++) {
    pha = convert(F[k], oldiphase, k, N); // updates oldiphase
    pha = harmonic_shift(pha, fm[b]);

    // ignore imag part
    val = unconvert(pha, oldophase, N); // updates oldophase
    Re(F[k]) = val;
    NEXT(F);
  }

  inphase[k] = oldiphase;
  outphase[k] = oldophase;
}


// blocks a multiple of 32 threads
// more blocks than multiprocessors
// perhaps 128 threads per block, as long as you have more blocks than
// multiprocessors
void threadblock(int device, int n, int *nblocks, int *nthreads)
{
  int nt, nb, nproc;
  hipDeviceProp_t deviceProp;

  //printf("%d threads needed\n", n);

  hipGetDeviceProperties(&deviceProp, device);
  nt = deviceProp.warpSize; // usually 32 threads
  nproc = deviceProp.multiProcessorCount;

  nb = 1 + (n - 1)/nt;

  
  //more blocks than procs
  if (nb >= 2*nproc) {
    nt *= 2; // 64 threads
    nb = 1 + (n - 1)/nt;
  }
   
  // // still more blocks than procs
  if (nb >= 2*nproc) {
    nt *= 2; // 128 threads
    nb = 1 + (n - 1)/nt;
  }

  if (nblocks) *nblocks = nb;
  if (nthreads) *nthreads = nt;  
}

typedef struct _SPV {
  OPDS h;
  MYFLT *out, *in, *shift, *iN;
  int nbins, ptr, nbtch, nblocks, nblocks1, nthreads, nthreads1, rblocks, rthreads;
  real *deltas, *ddeltas, *dinphase, *doutphase, *dfm;
  real *sine, *cosine;
  complex *dFin, *dFout, *dFoutw;
  real *samples, srate;
  real *framesin, *framesout, *fm;
  AUXCH aframesfm, aframesout, aframesin;
  uint32_t count;
} SPV;


void init_tables(CSOUND *csound, SPV *p, int nbins)
{
  int k;
  real s[nbins], c[nbins];

  for (k = 0; k < nbins; k++) {
    sincos(TWO_PI*k/nbins, &s[k], &c[k]);
  }

  CUDA(hipMemcpy, p->sine, s, nbins*sizeof(real), hipMemcpyHostToDevice);
  CUDA(hipMemcpy, p->cosine, c, nbins*sizeof(real), hipMemcpyHostToDevice);
}

void cuinit(CSOUND *csound, 
            SPV *p, uint32_t nframes, 
            int nbins, int device)
{
  int csize, rsize, off, nbins2;
  real binbw, nyq;
  hipDeviceProp_t deviceProp;

  if(!p->aframesin.auxp || p->aframesin.size < sizeof(real)*NBATCH) 
    csound->AuxAlloc(csound, sizeof(real)*NBATCH, &p->aframesin);
  if(!p->aframesout.auxp || p->aframesout.size < sizeof(real)*NBATCH) 
    csound->AuxAlloc(csound, sizeof(real)*NBATCH, &p->aframesout);
  if(!p->aframesfm.auxp || p->aframesfm.size < sizeof(real)*NBATCH) 
    csound->AuxAlloc(csound, sizeof(real)*NBATCH, &p->aframesfm);

  p->framesin = (real *) p->aframesin.auxp;
  p->framesout = (real *) p->aframesout.auxp;
  p->fm = (real *) p->aframesfm.auxp;

  p->nbins = nbins;
  nbins2 = nbins/2;
  p->nbtch = nframes;
  p->srate = (real) csound->GetSr(csound);
  binbw = p->srate/p->nbins;
  nyq = p->srate/real(2.0);

  CUDA(hipSetDevice, device);

  hipGetDeviceProperties(&deviceProp, device);
  
  // allocate contiguous arrays to aid hipMemcpy; round up to align
  // arrays for coalescence
  csize = roundup(p->nbins*sizeof(complex));
  rsize = roundup(p->nbins*sizeof(real));
  off = csize/sizeof(complex);

  if (off != rsize/sizeof(real)) {
    csound->InitError(csound, "Something weird happening with offsets at line %d\n",
		      __LINE__);
  }

  if (!csound->QueryGlobalVariable(csound, "::cusliding::init")) {
    csound->CreateGlobalVariable(csound, "::cusliding::init",1);
#ifndef RB
    csound->Message(csound, "Sliding PV: using floats on device %s (capability %d.%d)\n", deviceProp.name,
		    deviceProp.major, deviceProp.minor);
#else
    csound->Message(csound, "Sliding PV: using doubles on device %s (capability %d.%d)\n", deviceProp.name,
		    deviceProp.major, deviceProp.minor); 
#endif
    // global constants
    CUDA(hipMemcpyToSymbol, offset, &off, sizeof(int));
    CUDA(hipMemcpyToSymbol, nbatch, &p->nbtch, sizeof(int));
    CUDA(hipMemcpyToSymbol, binbandwidth, &binbw, sizeof(real));
    CUDA(hipMemcpyToSymbol, nyquist, &nyq, sizeof(real));

  }
  // N complexes
  CUDA(hipMalloc, &p->dFin, csize);
  CUDA(hipMemset, p->dFin, 0, csize);

  // N*nbatch complexes
  CUDA(hipMalloc, &p->dFout, p->nbtch*csize);
  CUDA(hipMalloc, &p->dFoutw, p->nbtch*csize);

  // N reals
  CUDA(hipMalloc, &p->dinphase, p->nbins*sizeof(real));
  CUDA(hipMalloc, &p->doutphase, p->nbins*sizeof(real));
  CUDA(hipMemset, p->dinphase, 0, p->nbins*sizeof(real));
  CUDA(hipMemset, p->doutphase, 0, p->nbins*sizeof(real));

  CUDA(hipMalloc, &p->sine, p->nbins*sizeof(real));
  CUDA(hipMalloc, &p->cosine, p->nbins*sizeof(real));

  // nbatch reals
  CUDA(hipMalloc, &p->ddeltas, p->nbtch*sizeof(real));
  CUDA(hipMalloc, &p->dfm, p->nbtch*sizeof(real));

  // circular history of nbins samples
  p->samples = (real*) csound->Calloc(csound, p->nbins*sizeof(real));
  if (p->samples == NULL) {
    csound->InitError(csound, "calloc failed at line %d\n", __LINE__);
    return;
  }

  // pinned memory is marginally faster
  //CUDA(hipHostAlloc, &deltas, p->nbtch*sizeof(real), hipHostMallocDefault);
  p->deltas = (real*) csound->Malloc(csound, p->nbtch*sizeof(real));
  if (p->deltas == NULL) {
    csound->InitError(csound, "malloc failed at line %d\n", __LINE__);
    return;
  }

  init_tables(csound, p, p->nbins);

  // index of oldest sample
  p->ptr = 0;
  
#ifdef RB
  threadblock(device, p->nbtch, &p->rblocks, &p->rthreads);
#else
  threadblock(device, p->nbtch*(nbins2), &p->rblocks, &p->rthreads);
  threadblock(device, p->nbtch*nbins, &p->nblocks1, &p->nthreads1);
#endif
  threadblock(device, (nbins2+1), &p->nblocks, &p->nthreads);

  if(csound->GetDebug(csound)) {
    csound->Message(csound, "%d bins, %d actual\n", p->nbins, nbins2 + 1);
    csound->Message(csound, "csize = %d -> %d bytes, offset %d elements\n", (int) (p->nbins*sizeof(complex)),
		    csize, off);
    csound->Message(csound, "%gHz bin bandwidth, Nyquist %gHz\n", binbw, nyq);
    csound->Message(csound, "period %d samples (%g sec)\n", p->nbtch, (real)p->nbtch/p->srate);
    csound->Message(csound, "%d multiprocessors\n", deviceProp.multiProcessorCount);
    csound->Message(csound, "%d blocks with %d threads = %d\n", p->nblocks, p->nthreads,
		    p->nblocks1*p->nthreads);
    csound->Message(csound, "%d idle threads in last block\n", p->nblocks*p->nthreads - 
		    (nbins2 + 1));
    csound->Message(csound, "%d blocks per SM\n",
		    (int)ceil((double)p->nblocks/deviceProp.multiProcessorCount));
    csound->Message(csound, "reconstruct using %d blocks of %d threads = %d\n", p->rblocks, p->rthreads,
		    p->rblocks*p->rthreads);
  }
}

int cushutdown(CSOUND *csound, void *pp)
{
  SPV *p = (SPV *) pp;
  csound->Free(csound, p->samples);

  CUDA(hipFree, p->dinphase);
  CUDA(hipFree, p->doutphase);
  CUDA(hipFree, p->ddeltas);
  CUDA(hipFree, p->dFoutw);
  CUDA(hipFree, p->dFout);
  CUDA(hipFree, p->dFin);
  CUDA(hipFree, p->sine);
  CUDA(hipFree, p->cosine);

  //CUDA(hipHostFree, deltas);
  csound->Free(csound, p->deltas);

  if(csound->GetDebug(csound))
    csound->Message(csound, "cuda shutdown\n");

  return OK;
}

// in samples with channels muxed 0 1 2 0 1 2 ...
void cuprocess(CSOUND *csound, SPV *p, real in[/*nbatch*/], real out[/*nbatch*/],
	       real fm[/*nbatch*/])
{
  int b;

  for (b = 0; b < p->nbtch; b++) {
    p->deltas[b] = in[b] - p->samples[p->ptr];
    p->samples[p->ptr] = in[b];
    p->ptr = (p->ptr + 1) % (p->nbins);
  }

  CUDA(hipMemcpy, p->ddeltas, p->deltas, p->nbtch*sizeof(real),
       hipMemcpyHostToDevice);

  slide<<<p->nblocks,p->nthreads>>>(p->ddeltas, p->dFin, p->dFoutw, 
				    p->sine, p->cosine, p->nbins);
  hipDeviceSynchronize(); CHK(slide);

  // Foutw and Fout must be separate to avoid data race
#ifdef RB
  window<<<p->nblocks,p->nthreads>>>(p->dFoutw, p->dFout, p->nbins);
#else
  window<<<p->nblocks1,p->nthreads1>>>(p->dFoutw, p->dFout, p->nbins);
#endif
  hipDeviceSynchronize(); CHK(window);

  CUDA(hipMemcpy, p->dfm, fm, p->nbtch*sizeof(real), hipMemcpyHostToDevice);

  fmsyn<<<p->nblocks,p->nthreads>>>(p->dinphase, p->dFout, p->dfm, p->doutphase, p->nbins);

  // reuse ddeltas array
  reconstruct<<<p->rblocks,p->rthreads>>>(p->dFout, p->ddeltas, p->nbins);
  CHK(reconstruct);
  // is is faster to transfer to pinned memory then copy; or just
  // transfer directly to out?
  CUDA(hipMemcpy, out, p->ddeltas, p->nbtch*sizeof(real),
       hipMemcpyDeviceToHost);

#ifndef RB
  for(b=0; b < p->nbtch; b++) out[b] /= p->nbins;
#endif

}


int spv_init(CSOUND *csound, SPV *p) {
  cuinit(csound,p,NBATCH,*p->iN,0);
  csound->RegisterDeinitCallback(csound, p, cushutdown);
  return OK;
}

int spv_perf(CSOUND *csound, SPV *p) {
  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t n, nsmps = CS_KSMPS;
  uint32_t count = p->count;

  if (UNLIKELY(offset)) memset(p->out, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&p->out[nsmps], '\0', early*sizeof(MYFLT));
  }

  for(n=offset; n < nsmps; n++) {
    p->framesin[count] = p->in[n];
    p->out[n] = p->framesout[count];
    p->fm[count] = p->shift[n];
    count++;
    if(count == NBATCH) {
      cuprocess(csound, p,p->framesin,p->framesout,p->fm);
      count = 0;
    }
    
  }

  p->count = count;
  return OK;
}

static OENTRY localops[] = {
  {"cudasliding", sizeof(SPV),0, 5, "a", "aai", (SUBR) spv_init, NULL,
   (SUBR) spv_perf}
};

extern "C" {
  LINKAGE
}



