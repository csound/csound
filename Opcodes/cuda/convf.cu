#include "hip/hip_runtime.h"
// -*- c++ -*-
/* convf.cu
  (c) Victor Lazzarini, 2013

  based on M Puckette's pitch tracking algorithm.

  This file is part of Csound.

  The Csound Library is free software; you can redistribute it
  and/or modify it under the terms of the GNU Lesser General Public
  License as published by the Free Software Foundation; either
  version 2.1 of the License, or (at your option) any later version.

  Csound is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU Lesser General Public License for more details.

  You should have received a copy of the GNU Lesser General Public
  License along with Csound; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA
  02111-1307 USA
*/

#include <csdl.h>

__global__ void convol(float *out, float *del, float *coefs, int irsize, int rp, int vsize) {
  int t = (threadIdx.x + blockIdx.x*blockDim.x);
  if(t >= irsize*vsize) return;
  int n =  t%vsize;  /* sample index */
  int h =  t/vsize;  /* coeff index */
  int end = irsize+vsize;
  rp += n + h; /* read point, oldest -> newest */
  out[t] = del[rp < end ? rp : rp%end]*coefs[irsize-1-h];  /* single tap */
  if(t >= vsize) return;
  syncthreads();
  float a = 0.0;
  for(int i=1, j=vsize; i < irsize; i++, j+=vsize)
    a +=  out[n + j]; /* mix all taps */   
  out[n] += a;    
}

typedef struct _CONV {
  OPDS h;
  MYFLT *aout, *asig, *ifn;
  float *coeffs, *out, *del;
  int wp, irsize;
  AUXCH buf;
  int blocks, threads;
} CONV;


static int destroy_conv(CSOUND *csound, void *pp){
  CONV *p = (CONV *) pp;
  hipFree(p->coeffs);
  hipFree(p->del);
  hipFree(p->out);
  return OK;
}

static int conv_init(CSOUND *csound, CONV *p){

  FUNC *ftab = csound->FTnp2Find(csound, p->ifn);
  int irsize = ftab->flen;
  int nsmps = CS_KSMPS,i;
  int threads = irsize*nsmps;
  float *tmp;

  hipMalloc(&p->coeffs, sizeof(float)*irsize);   

  tmp = (float*) malloc(sizeof(float)*irsize);
  for(i=0; i< irsize; i++)
    tmp[i] = (float) ftab->ftable[i];
  hipMemcpy(p->coeffs,tmp, sizeof(float)*irsize, 
            hipMemcpyHostToDevice); 
  free(tmp);
   
  hipMalloc(&p->del, sizeof(float)*(irsize+nsmps)); 
  hipMalloc(&p->out, sizeof(float)*threads); 
  hipMemset(p->del,0,sizeof(float)*(irsize+nsmps));
  hipMemset(p->out, 0, sizeof(float)*threads);
  
  p->wp = 0;
  p->irsize = irsize;

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  int blockspt = deviceProp.maxThreadsPerBlock;
  csound->Message(csound, "CUDAconv: using device %s (capability %d.%d)\n", 
        deviceProp.name,deviceProp.major, deviceProp.minor);

  p->blocks = threads > blockspt ? ceil(threads/blockspt) : 1;
  p->threads = threads > blockspt ? blockspt : threads;

  csound->RegisterDeinitCallback(csound, p, destroy_conv);
  OPARMS parms;
  csound->GetOParms(csound, &parms);
  if(parms.odebug)
   csound->Message(csound, "blocks %d, threads %d - %d\n", p->blocks, p->threads, threads);
  if(p->buf.auxp == NULL)
    csound->AuxAlloc(csound, sizeof(float)*CS_KSMPS, &p->buf);

  return OK;

}
/* the delay size is irsize + vsize so that
   we can shift in a whole block of samples */
int conv_perf(CSOUND *csound, CONV *p){

   int nsmps = CS_KSMPS;
   MYFLT *sig = p->asig, *aout = p->aout;
   float *del = p->del, *out = p->out, *coefs = p->coeffs, *buf = (float *)p->buf.auxp;
   int irsize = p->irsize;
   int wp = p->wp, i;
  
  for(i=0; i < nsmps; i++) buf[i] = (float) sig[i]; 
  if(wp > irsize) {
     int front = wp - irsize; 
     hipMemcpy(&del[wp], buf, sizeof(float)*(nsmps-front), hipMemcpyHostToDevice);
     hipMemcpy(del, &buf[nsmps-front], sizeof(float)*front, hipMemcpyHostToDevice);
  } 
  else hipMemcpy(&del[wp], buf, sizeof(float)*nsmps, hipMemcpyHostToDevice); 
  
  wp = (wp+nsmps)%(irsize+nsmps); /* wp is now the oldest sample in the delay */
  convol<<<p->blocks,p->threads>>>(out, del, coefs, irsize, wp, nsmps);

  hipMemcpy(buf, out, sizeof(float)*nsmps, hipMemcpyDeviceToHost); 

  for(i=0; i < nsmps; i++) aout[i] = (float) buf[i]; 
  p->wp = wp;
  return OK;
}

static OENTRY localops[] = {
  {"cudaconv", sizeof(CONV),0, 5, "a", "ai", (SUBR) conv_init, NULL,
   (SUBR) conv_perf},
};

extern "C" {
  LINKAGE
}
