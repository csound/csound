#include "hip/hip_runtime.h"
// -*- c++ -*-
// adsyn.cu
// experimental cuda opcodes
//
// V Lazzarini, 2013

#include <csdl.h>
#include <hipfft/hipfft.h>
#define VSAMPS 64

//__shared__ MYFLT mema[64*20];
#define PFRACLO(x)   ((MYFLT)((x) & lomask) * lodiv)

__global__ void component_table(MYFLT *out, int64_t *ndx, MYFLT *tab,
                          float *amp, int *inc, int vsize,
                          int blocks, int lobits, MYFLT lodiv,
                          int lomask) {

  int h = threadIdx.x*blocks + blockIdx.x;
  int i, offset, n;
  int64_t lndx;
  offset = h*vsize;
  out += offset;

  for(i=0; i < vsize; i++) {
    lndx = ndx[h];
    n = lndx >> lobits;
    out[i] = amp[h]*(tab[n] +  PFRACLO(lndx)*(tab[n+1] - tab[n]));
    ndx[h] = (lndx + inc[h]) & PHMASK;
  }

}

__global__ void component_sine(MYFLT *out, int64_t *ndx,
                          float *amp, int *inc, int vsize,
                          int blocks) {

  int h = threadIdx.x*blocks + blockIdx.x;
  int i, offset;
  int64_t lndx;
  offset = h*vsize;
  out += offset;

  for(i=0; i < vsize; i++) {
    lndx = ndx[h];
    out[i] = amp[h]*sin((PI*2*lndx)/FMAXLEN);
    ndx[h] = (lndx + inc[h]) & PHMASK;
  }

}

__global__  void mixdown_(MYFLT *out, int comps, int vsize, float kamp){
   int h = threadIdx.x;
   int i;
   for(i=1; i < comps; i++){
     out[h] +=  out[h + vsize*i];
   }
   out[h] *= kamp;
}


static int destroy_cudaop(CSOUND *csound, void *pp);

typedef struct cudaop_ {
  OPDS h;
  MYFLT *asig;
  MYFLT *kamp, *kfreq, *itabn;
  MYFLT *ftabn, *atabn, *inum;
  MYFLT *out;
  float *amp;
  MYFLT *tab;
  int64_t *ndx;
  int *inc;
  MYFLT *ap, *fp;
  FUNC *itab, *ftab, *atab;
  int N, blocks;
} CUDAOP;

static int init_cudaop(CSOUND *csound, CUDAOP *p){

  int a, b, asize, ipsize, fpsize, tsize;
  int nsmps = CS_KSMPS, blockspt;
  if(nsmps > 1024) return csound->InitError(csound, "ksmps is too large\n");
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  blockspt = deviceProp.maxThreadsPerBlock;
  if(deviceProp.major < 3)
    csound->InitError(csound,
     "this opcode requires device capability 3.0 minimum\n");

  if(*p->itabn != 0){
  if((p->itab =
      csound->FTFind(csound, p->itabn))== NULL)
    return csound->InitError(csound,
                             "could not find table %.0f\n", *p->itabn);
  } else p->itab = NULL;

  if((p->ftab =
      csound->FTnp2Find(csound, p->ftabn))== NULL)
    return csound->InitError(csound,
                             "could not find table %.0f\n", *p->ftabn);

  if((p->atab =
      csound->FTnp2Find(csound, p->atabn))== NULL)
    return csound->InitError(csound,
                             "could not find table %.0f\n", *p->atabn);

  a = p->ftab->flen;
  b = p->atab->flen;
  p->N = a < b ? a : b;

  if(*p->inum > 0 && *p->inum < p->N) p->N = *p->inum;

  p->blocks = p->N > blockspt ? p->N/blockspt : 1;

  asize = p->N*nsmps*sizeof(MYFLT);
  ipsize = p->N*sizeof(int64_t);
  fpsize = p->N*sizeof(float);
  if(p->itab)
   tsize = (p->itab->flen+1)*sizeof(MYFLT);

  hipMalloc(&p->out, asize);
  hipMalloc(&p->ndx, ipsize);
  hipMalloc(&p->amp, fpsize);
  hipMalloc(&p->inc, ipsize);
  if(p->itab) {
   hipMalloc(&p->tab, tsize);
   hipMemcpy(p->tab, p->itab->ftable, tsize, hipMemcpyHostToDevice);
  }
  hipMemset(p->ndx, 0, ipsize);

  p->ap = p->atab->ftable;
  p->fp = p->ftab->ftable;

  csound->RegisterDeinitCallback(csound, p, destroy_cudaop);
  csound->Message(csound, "%d threads, %d blocks\n", p->N, p->blocks);
  return OK;
}

static void update_params(CSOUND *csound, CUDAOP *p){

  int ipsize = p->N*sizeof(int);
  int fpsize = p->N*sizeof(float);
  float amp[p->N];
  int inc[p->N], i;
  int N = p->N;

   for(i=0;i < N; i++){
    amp[i] = p->ap[i];
    inc[i] = *p->kfreq*p->fp[i]*FMAXLEN/csound->GetSr(csound);
   }
   hipMemcpy(p->amp,amp,fpsize, hipMemcpyHostToDevice);
   hipMemcpy(p->inc,inc,ipsize, hipMemcpyHostToDevice);

}

static int perf_cudaop(CSOUND *csound, CUDAOP *p){

  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t nsmps = CS_KSMPS;
  p->ap = p->atab->ftable;
  p->fp = p->ftab->ftable;

  if (UNLIKELY(offset)) memset(p->asig, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&(p->asig[nsmps]), '\0', early*sizeof(MYFLT));
  }

  update_params(csound, p);
  if(p->itab)
   component_table<<<p->blocks,
        p->N/p->blocks>>>(p->out,p->ndx,
                          p->tab,p->amp,
                          p->inc,nsmps,
                          p->blocks,
                          p->itab->lobits,
                          p->itab->lodiv,
                          p->itab->lomask);
  else
   component_sine<<<p->blocks,
        p->N/p->blocks>>>(p->out,p->ndx,
                          p->amp,
                          p->inc,nsmps,
                          p->blocks);
   mixdown_<<<1,nsmps>>>(p->out,p->N,nsmps,*p->kamp);
   hipMemcpy(p->asig,p->out,nsmps*sizeof(MYFLT),hipMemcpyDeviceToHost);

  return OK;
}

static int destroy_cudaop(CSOUND *csound, void *pp){
  CUDAOP *p = (CUDAOP *) pp;
  hipFree(p->out);
  hipFree(p->ndx);
  hipFree(p->tab);
  hipFree(p->amp);
  hipFree(p->inc);
  return OK;
}


#include <pstream.h>

typedef struct cudaop2_ {
  OPDS h;
  MYFLT *asig;
  PVSDAT *fsig;
  MYFLT *kamp, *kfreq;
  MYFLT *inum;
  MYFLT *out;
  float *frame;
  int64_t *ndx;
  float *fp, *previous;
  AUXCH out_;
  int bins, blocks, threads;
  int count;
  int vsamps, mblocks, mthreads;
  int framecount;
} CUDAOP2;

static int destroy_cudaop2(CSOUND *csound, void *pp);

static int init_cudaop2(CSOUND *csound, CUDAOP2 *p){

  int asize, ipsize, fpsize, blockspt;
  if(p->fsig->overlap > 1024)
     return csound->InitError(csound, "overlap is too large\n");
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  blockspt = deviceProp.maxThreadsPerBlock;
  if(deviceProp.major < 3)
    csound->InitError(csound,
   "this opcode requires device capability 3.0 minimum\n");

  p->bins = (p->fsig->N)/2;

  if(*p->inum > 0 && *p->inum < p->bins) p->bins = *p->inum;

  p->vsamps = p->fsig->overlap;
  p->threads = p->bins*p->vsamps;
  p->blocks = p->threads > blockspt ? p->threads/blockspt : 1;
  p->mthreads = p->bins > p->vsamps ? p->bins : p->vsamps;
  p->mblocks = p->mthreads >  blockspt ? p->mthreads/blockspt : 1;

  p->threads /= p->blocks;
  p->mthreads /= p->mblocks;

  asize = p->bins*p->vsamps*sizeof(MYFLT);
  ipsize  =p->fsig->N*sizeof(int64_t)/2;
  fpsize = p->fsig->N*sizeof(float)*2;

  hipMalloc(&p->out, asize);
  hipMalloc(&p->ndx, ipsize);
  hipMalloc(&p->frame, fpsize);
  hipMalloc(&p->previous, fpsize);
  hipMemset(p->previous, 0, fpsize);
  hipMemset(p->ndx, 0, ipsize);

  asize = p->vsamps*sizeof(MYFLT);
  if(p->out_.auxp == NULL ||
     p->out_.size < asize)
    csound->AuxAlloc(csound, asize , &p->out_);

  csound->RegisterDeinitCallback(csound, p, destroy_cudaop2);
  p->count = 0;
  return OK;
}

//__shared__ int64_t ph[2048];

__global__ void sample(MYFLT *out, float *frame, MYFLT pitch, int64_t *ph,
                       float *amps, int bins, int vsize, MYFLT sr) {

  int t = (threadIdx.x + blockIdx.x*blockDim.x);
  int n =  t%vsize;  /* sample index */
  int h = t/vsize;  /* bin index */
  int k = h<<1;
  int64_t lph;
  float a = amps[h], ascl = ((float)n)/vsize;
  MYFLT fscal = pitch*FMAXLEN/sr;
  lph = (ph[h] + (int64_t)(n*round(frame[k+1]*fscal))) & PHMASK;
  a += ascl*(frame[k] - a);
  out[t] = a*sinf((2*PI*lph)/FMAXLEN);
}

__global__ void updatemix(MYFLT *out, float *frame, float *amps, MYFLT kamp,
           int64_t *ph, MYFLT pitch, int bins, int vsize, MYFLT sr){

 int h = threadIdx.x + blockIdx.x*blockDim.x;
 int k = h << 1, i;
 /* update phases and amps */
 ph[h]  = (ph[h] + (int64_t)(vsize*round(pitch*frame[k+1]*FMAXLEN/sr))) & PHMASK;
 amps[h] = frame[k];
 if(h > vsize) 
   return;
 /* mix all partials */
  for(i=1; i < bins; i++){
    out[h] +=  out[h + vsize*i];
  }
 out[h] *= kamp;
}


static int perf_cudaop2(CSOUND *csound, CUDAOP2 *p){

  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t n, nsmps = CS_KSMPS;
  MYFLT *out_ = (MYFLT *) p->out_.auxp;
  MYFLT      *asig = p->asig;
  int count = p->count,  vsamps = p->vsamps;
  p->fp = (float *) (p->fsig->frame.auxp);

  if (UNLIKELY(offset)) memset(asig, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&asig[nsmps], '\0', early*sizeof(MYFLT));
   }

  for(n=offset; n < nsmps; n++){
    if(count == 0) {
      hipMemcpy(p->frame,p->fp,sizeof(float)*p->bins*2,hipMemcpyHostToDevice);
      sample<<<p->blocks,p->threads>>>(p->out,p->frame,
                                               *p->kfreq,
                                                p->ndx,
                                                p->previous,
                                                p->bins,
                                                vsamps,
                                                csound->GetSr(csound));
      // if (hipDeviceSynchronize() != hipSuccess)
      // csound->Message(csound,"Cuda error: Failed to synchronize\n");
      updatemix<<<p->mblocks,p->mthreads>>>(p->out, p->frame,
                                            p->previous, *p->kamp,
                                            p->ndx,
                                            *p->kfreq,
                                            p->bins,
                                            vsamps,
                                            csound->GetSr(csound));
      hipMemcpy(out_,p->out,vsamps*sizeof(MYFLT),hipMemcpyDeviceToHost);
      count = vsamps;
    }
    asig[n] = (MYFLT) out_[vsamps - count];
    count--;
  }
  p->count = count;
  return OK;
}

static int destroy_cudaop2(CSOUND *csound, void *pp){
  CUDAOP2 *p = (CUDAOP2 *) pp;
  hipFree(p->out);
  hipFree(p->ndx);
  hipFree(p->frame);
  return OK;
}


static OENTRY localops[] = {
  {"cudasynth", sizeof(CUDAOP),0, 5, "a", "kkiiio", (SUBR) init_cudaop, NULL,
   (SUBR) perf_cudaop},
  {"cudasynth", sizeof(CUDAOP2),0, 5, "a", "fkko", (SUBR) init_cudaop2, NULL,
   (SUBR) perf_cudaop2}
};

extern "C" {
  LINKAGE
}
