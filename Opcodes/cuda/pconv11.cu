#include "hip/hip_runtime.h"
// -*- c++ -*-
/* pconv.cu
  experimental cuda opcodes
  (c) Victor Lazzarini, 2013

  based on M Puckette's pitch tracking algorithm.

  This file is part of Csound.

  The Csound Library is free software; you can redistribute it
  and/or modify it under the terms of the GNU Lesser General Public
  License as published by the Free Software Foundation; either
  version 2.1 of the License, or (at your option) any later version.

  Csound is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU Lesser General Public License for more details.

  You should have received a copy of the GNU Lesser General Public
  License along with Csound; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA
  02111-1307 USA
*/

#include <csdl.h>
#include <hipfft/hipfft.h>

/* 
  each kernel processes one bin 
*/
__global__ void pconvol(float *out,float *in, 
			float *coef,int rp, int dftsize, 
			int nparts, int end) {
  float re,im,re2,im2;   

  /* thread count */
  int t = (threadIdx.x + blockIdx.x*blockDim.x);

  int k = t<<1;       /* coef pos      */   
  int n = k%(dftsize+2);  /* inframe pos   */

  /* if beyond the buffer end, exit */
  if(k >= end) return;                  
  rp += k/(dftsize+2);       /*  rp pos */

  /* select correct input buffer */
  in += (rp < nparts ? rp : rp%nparts)*(dftsize+2);

  re = coef[k]; im = coef[k+1];
  re2 = in[n];  im2 = in[n+1];
  
  /* complex multiplication + sums */
  out[k] = re*re2 - im*im2;
  out[k+1] = re*im2 + re2*im;

  if(t > dftsize+1) return;
  syncthreads();
    for(int i=dftsize+2; i < end; i+=(dftsize+2))
      out[t] += out[t + i];

 
}  

/* sample-by-sample overlap-save operation */
__global__ void olapsave(float *buf, float *in, int parts){
   int n = (threadIdx.x + blockIdx.x*blockDim.x);
   buf[n] = in[n] + buf[parts+n];
   buf[parts+n] = in[parts+n];
}



typedef struct _pconv{
  OPDS h;
  MYFLT *aout, *asig, *ifn, *parts;
  float *out, *coef, *in, *buf;
  AUXCH  bufin, bufout;
  int wp, nparts, dftsize, cnt;
  hipfftHandle plan, iplan;
  int threads, blocks, othreads, oblocks;
} PCONV;


int isPowerOfTwo (unsigned int x)
{
  return ((x != 0) && !(x & (x - 1)));
}


static int destroy_pconv(CSOUND *csound, void *pp){
  PCONV *p = (PCONV *) pp;
  hipfftDestroy(p->plan);
  hipfftDestroy(p->iplan);
  hipFree(p->coef);
  hipFree(p->in);
  hipFree(p->out);
  hipFree(p->buf);
  return OK;
}


int pconv_init(CSOUND *csound, PCONV *p){

  FUNC *ftab = csound->FTnp2Find(csound, p->ifn);
  float *tmp;
  int tlen = ftab->flen;
  int end, i, j, k, parts = *p->parts, dftsize, nparts;
  MYFLT *tab = ftab->ftable;

  if(!isPowerOfTwo(parts))
    return csound->InitError(csound, "partition size needs to be power of two\n");

  if(parts > tlen)
     return csound->InitError(csound, "partition size too big \n");

  end = tlen + parts - 1;

  nparts = end / parts;
  dftsize = parts << 1;
  end = nparts*(dftsize+2);

  hipMalloc(&p->coef, sizeof(float)*end);   
  hipMalloc(&p->in, sizeof(float)*end); 
  hipMalloc(&p->out, sizeof(float)*end); 
  hipMalloc(&p->buf, sizeof(float)*(dftsize));
  
  hipMemset(p->in,0,sizeof(float)*end);
  hipMemset(p->out, 0, sizeof(float)*end);
  hipMemset(p->buf, 0, sizeof(float)*(dftsize));
  hipMemset(p->coef, 0, sizeof(float)*end);

  p->wp = 0;

  if(!p->bufin.auxp || p->bufin.size < sizeof(float)*dftsize)
     csound->AuxAlloc(csound, sizeof(float)*dftsize, &p->bufin);
  if(!p->bufout.auxp || p->bufout.size < sizeof(float)*parts)
     csound->AuxAlloc(csound, sizeof(float)*parts, &p->bufout);

  memset(p->bufout.auxp, 0, sizeof(float)*parts);

    hipfftResult res;

  tmp = (float *) p->bufin.auxp;
  if((res = hipfftPlan1d(&p->plan, dftsize, HIPFFT_R2C, 1))
  != HIPFFT_SUCCESS) csound->Message(csound, "cuda fft setup error %d\n", res);
  cufftSetCompatibilityMode(p->plan, CUFFT_COMPATIBILITY_NATIVE);
  hipfftPlan1d(&p->iplan, dftsize, HIPFFT_C2R, 1);
  cufftSetCompatibilityMode(p->iplan, CUFFT_COMPATIBILITY_NATIVE);


  for(i =0, k=0; i < nparts; i++){
    for(j=0; j < dftsize; j++)
      tmp[j] = j < parts && k < tlen ? tab[k++] : 0.f;
      float *pp = p->coef + (nparts - 1 - i)*(dftsize+2);
    hipMemcpy(pp, tmp, sizeof(float)*dftsize, 
               hipMemcpyHostToDevice); 
    if((res = hipfftExecR2C(p->plan,pp,(hipfftComplex*)pp))
       != HIPFFT_SUCCESS) csound->Message(csound, "cuda fft error %d\n", res);
   }

  hipDeviceSynchronize();
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  int blockspt = deviceProp.maxThreadsPerBlock;
 
  end >>= 1;

  p->blocks = end > blockspt ? ceil(end/blockspt) : 1;
  p->threads = end > blockspt ? blockspt : end;
  p->oblocks = parts > blockspt ? ceil(parts/blockspt) : 1;
  p->othreads = parts > blockspt ? blockspt : parts;

  csound->RegisterDeinitCallback(csound, p, destroy_pconv);

  OPARMS parms;
  csound->GetOParms(csound, &parms);
  if(parms.odebug)
   csound->Message(csound, 
     "blocks %d - threads/block %d - threads %d - dftsize %d - parts %d\n", 
		   p->blocks, p->threads, end, dftsize, nparts);

  p->nparts = nparts;
  p->dftsize = dftsize;
  p->cnt = 0;

  return OK;
}

int pconv_perf(CSOUND *csound, PCONV *p){

  int dftsize = p->dftsize, cnt = p->cnt, wp = p->wp, nparts = p->nparts;
  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t n, nsmps = CS_KSMPS;
  float *bufin = (float *) p->bufin.auxp, *bufout = (float *) p->bufout.auxp;
  MYFLT *asig = p->asig, *aout = p->aout;
  float *in = p->in, *out = p->out, *coef = p->coef, *buf = p->buf;
  int end = nparts*(dftsize+2);
  int parts = *p->parts;

  if (UNLIKELY(offset)) memset(asig, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&asig[nsmps], '\0', early*sizeof(MYFLT));
  }

  for(n = offset; n < nsmps; n++){
    bufin[cnt] = (float) asig[n];
    aout[n] = (MYFLT) bufout[cnt]/dftsize;
  
    if(++cnt == parts){
       /* in buffer pos */
      int pos = wp*(dftsize+2);

       /* increment delay line pos
          so that it points to the oldest partition
       */
       wp += 1;
       if(wp == nparts) wp = 0;

       /* copy current buffer into newest partition */
       hipMemset(out, 0, sizeof(float)*(dftsize+2));
       hipMemcpy(&in[pos],bufin,sizeof(float)*dftsize,hipMemcpyHostToDevice);

       /* apply transform */
       if(hipfftExecR2C(p->plan,&in[pos],(hipfftComplex*)&in[pos])
         != HIPFFT_SUCCESS) csound->Message(csound, "cuda in fft error\n");
       if (hipDeviceSynchronize() != hipSuccess)
        csound->Message(csound,"Cuda error: Failed to synchronize\n");

       /* convolution */
       pconvol<<<p->blocks,p->threads>>>(out, in, coef, wp, dftsize, nparts, end);

       if (hipDeviceSynchronize() != hipSuccess)
        csound->Message(csound,"Cuda error: Failed to synchronize\n");

       /* transform output */
       if(hipfftExecC2R(p->iplan,(hipfftComplex*)out,out) 
        != HIPFFT_SUCCESS) csound->Message(csound, "cuda out fft error\n");

       if (hipDeviceSynchronize() != hipSuccess)
        csound->Message(csound,"Cuda error: Failed to synchronize\n");

       /* overlap-save */
       olapsave<<<p->oblocks,p->othreads>>>(buf,out,parts); 
 
       /* copy buffer out */
       hipMemcpy(bufout,buf, sizeof(float)*parts,hipMemcpyDeviceToHost);

       cnt = 0;
    }
  }
  p->cnt = cnt;
  p->wp = wp;
  return OK;
}

static OENTRY localops[] = {
  {"cudapconv", sizeof(PCONV),0, 5, "a", "aii", (SUBR) pconv_init, NULL,
    (SUBR) pconv_perf},
};

extern "C" {
  LINKAGE
}
