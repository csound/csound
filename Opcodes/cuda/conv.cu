#include "hip/hip_runtime.h"
// -*- c++ -*-
// conv.cu
// experimental cuda opcodes
//
// V Lazzarini, 2014

#include <csdl.h>

__global__ void convol(MYFLT *out, MYFLT *del, MYFLT *coefs, int irsize, int rp, int vsize) {
  int t = (threadIdx.x + blockIdx.x*blockDim.x);
  if(t >= irsize*vsize) return;
  int n =  t%vsize;  /* sample index */
  int h =  t/vsize;  /* coeff index */
  int end = irsize+vsize;
  rp += n + h; /* read point, oldest -> newest */
  out[t] = del[rp < end ? rp : rp%end]*coefs[irsize-1-h];  /* single tap */
  if(t >= vsize) return;
  syncthreads();
  MYFLT a = 0.0;
  for(int i=1, j=vsize; i < irsize; i++, j+=vsize)
    a +=  out[n + j]; /* mix all taps */   
  out[n] += a;    
}

// __device__ double atomicAdd(double* address, double val)
// {
//     unsigned long long int* address_as_ull =
//                               (unsigned long long int*)address;
//     unsigned long long int old = *address_as_ull, assumed;
//     do {
//         assumed = old;
//         old = atomicCAS(address_as_ull, assumed,
//                         __double_as_longlong(val +
//                                __longlong_as_double(assumed)));
//     } while (assumed != old);
//     return __longlong_as_double(old);
// }

// __global__ void convol2(MYFLT *out, MYFLT *del, MYFLT *coefs, int irsize, int rp, int vsize) {
//   int t = (threadIdx.x + blockIdx.x*blockDim.x);
//   if(t >= irsize*vsize) return;
//   int n =  t%vsize;  /* sample index */
//   int h =  t/vsize;  /* coeff index */
//   int end = irsize+vsize;
//   rp += n + h; /* read point, oldest -> newest */
//   MYFLT s = del[rp < end ? rp : rp%end]*coefs[irsize-1-h];  /* single tap */
//   t == n ? out[n] = s : atomicAdd(&out[n], s);
// }

typedef struct _CONV {
  OPDS h;
  MYFLT *aout, *asig, *ifn;
  MYFLT *coeffs, *out, *del;
  int wp, irsize;
  int blocks, threads;
} CONV;


static int destroy_conv(CSOUND *csound, void *pp){
  CONV *p = (CONV *) pp;
  hipFree(p->coeffs);
  hipFree(p->del);
  hipFree(p->out);
  return OK;
}

static int conv_init(CSOUND *csound, CONV *p){

  FUNC *ftab = csound->FTnp2Find(csound, p->ifn);
  int irsize = ftab->flen;
  int nsmps = CS_KSMPS;
  int threads = irsize*nsmps;
 
  hipMalloc(&p->coeffs, sizeof(MYFLT)*irsize);   
  hipMemcpy(p->coeffs, ftab->ftable, sizeof(MYFLT)*irsize, 
            hipMemcpyHostToDevice); 
   
  hipMalloc(&p->del, sizeof(MYFLT)*(irsize+nsmps)); 
  hipMalloc(&p->out, sizeof(MYFLT)*threads); 
  hipMemset(p->del,0,sizeof(MYFLT)*(irsize+nsmps));
  hipMemset(p->out, 0, sizeof(MYFLT)*threads);
  
  p->wp = 0;
  p->irsize = irsize;

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  int blockspt = deviceProp.maxThreadsPerBlock;
  csound->Message(csound, "CUDAconv: using device %s (capability %d.%d)\n", 
        deviceProp.name,deviceProp.major, deviceProp.minor);

  p->blocks = threads > blockspt ? ceil(threads/blockspt) : 1;
  p->threads = threads > blockspt ? blockspt : threads;

  csound->RegisterDeinitCallback(csound, p, destroy_conv);
  OPARMS parms;
  csound->GetOParms(csound, &parms);
  if(parms.odebug)
   csound->Message(csound, "blocks %d, threads %d - %d\n", p->blocks, p->threads, threads);

  return OK;

}
/* the delay size is irsize + vsize so that
   we can shift in a whole block of samples */
int conv_perf(CSOUND *csound, CONV *p){

   int nsmps = CS_KSMPS;
   MYFLT *sig = p->asig, *aout = p->aout;
   MYFLT *del = p->del, *out = p->out, *coefs = p->coeffs;
   int irsize = p->irsize;
   int wp = p->wp;

  if(wp > irsize) {
     int front = wp - irsize; 
     hipMemcpy(&del[wp], sig, sizeof(MYFLT)*(nsmps-front), hipMemcpyHostToDevice);
     hipMemcpy(del, &sig[nsmps-front], sizeof(MYFLT)*front, hipMemcpyHostToDevice);
  } 
  else hipMemcpy(&del[wp], sig, sizeof(MYFLT)*nsmps, hipMemcpyHostToDevice); 
  
  wp = (wp+nsmps)%(irsize+nsmps); /* wp is now the oldest sample in the delay */
  convol<<<p->blocks,p->threads>>>(out, del, coefs, irsize, wp, nsmps);

  hipMemcpy(aout, out, sizeof(MYFLT)*nsmps, hipMemcpyDeviceToHost); 
  p->wp = wp;
  return OK;
}

static OENTRY localops[] = {
  {"cudaconv", sizeof(CONV),0, 5, "a", "ai", (SUBR) conv_init, NULL,
   (SUBR) conv_perf},
};

extern "C" {
  LINKAGE
}
