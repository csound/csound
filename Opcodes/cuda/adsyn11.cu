#include "hip/hip_runtime.h"
// -*- c++ -*-
/* adsyn11.cu
  (c) Victor Lazzarini, 2013

  based on M Puckette's pitch tracking algorithm.

  This file is part of Csound.

  The Csound Library is free software; you can redistribute it
  and/or modify it under the terms of the GNU Lesser General Public
  License as published by the Free Software Foundation; either
  version 2.1 of the License, or (at your option) any later version.

  Csound is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU Lesser General Public License for more details.

  You should have received a copy of the GNU Lesser General Public
  License along with Csound; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA
  02111-1307 USA
*/

#include <csdl.h>
#include <pstream.h>

typedef struct cudadsyn_ {
  OPDS h;
  MYFLT *asig;
  PVSDAT *fsig;
  MYFLT *kamp, *kfreq;
  MYFLT *inum;
  float *out;
  float *frame;
  int64_t *ndx;
  float *fp, *previous;
  AUXCH out_;
  int bins, blocks, threads;
  int count;
  int vsamps, mblocks, mthreads;
  int framecount;
} CUDADSYN;

static int destroy_cudadsyn(CSOUND *csound, void *pp);

static int init_cudadsyn(CSOUND *csound, CUDADSYN *p){

  int asize, ipsize, fpsize, blockspt;
  if(p->fsig->overlap > 1024)
     return csound->InitError(csound, "overlap is too large\n");
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  blockspt = deviceProp.maxThreadsPerBlock;
  if(deviceProp.major < 1)
   csound->InitError(csound,
		     "this opcode requires device capability 1.0 minimum. Device is %d.%d\n", 
        deviceProp.major, deviceProp.minor );

  p->bins = (p->fsig->N)/2;

  if(*p->inum > 0 && *p->inum < p->bins) p->bins = *p->inum;

  p->vsamps = p->fsig->overlap;
  p->threads = p->bins*p->vsamps;
  p->blocks = p->threads > blockspt ? p->threads/blockspt : 1;
  p->mthreads = p->bins;
  p->mblocks = p->mthreads >  blockspt ? p->mthreads/blockspt : 1;

  p->threads /= p->blocks;
  p->mthreads /= p->mblocks;

  asize =  p->vsamps*p->bins*sizeof(float);
  ipsize = p->fsig->N*sizeof(int64_t)/2;
  fpsize = p->fsig->N*sizeof(float);

  hipMalloc(&p->out, asize);
  hipMalloc(&p->ndx, ipsize);
  hipMalloc(&p->frame, fpsize);
  hipMalloc(&p->previous, fpsize/2);
  hipMemset(p->previous, 0, fpsize/2);
  hipMemset(p->ndx, 0, ipsize);

  asize = p->vsamps*sizeof(float);
  if(p->out_.auxp == NULL ||
     p->out_.size < asize)
    csound->AuxAlloc(csound, asize , &p->out_);

  csound->RegisterDeinitCallback(csound, p, destroy_cudadsyn);
  p->count = 0;
  return OK;
}

__global__ void sample(float *out, float *frame, float pitch, int64_t *ph,
                       float *amps, int bins, int vsize, float sr) {

  int t = (threadIdx.x + blockIdx.x*blockDim.x);
  int n =  t%vsize;  /* sample index */
  int h = t/vsize;  /* bin index */
  int k = h<<1;
  int64_t lph; 
  float a = amps[h], ascl = ((float)n)/vsize;
  float fscal = pitch*FMAXLEN/sr;
  lph = (ph[h] + (int64_t)(n*round(frame[k+1]*fscal))) & PHMASK;
  a += ascl*(frame[k] - a);
  out[t] = a*sinf((2*PI*lph)/FMAXLEN);
  if(t >= vsize) return;
  syncthreads();
  for(int i=vsize; i < vsize*bins; i+=vsize)
    out[t] += out[t + i];
}

__global__ void update(float *frame, float *amps,
      int64_t *ph,float pitch, int vsize, float sr){

 int h = threadIdx.x + blockIdx.x*blockDim.x;
 int k = h << 1;
 /* update phases and amps */
 ph[h]  = (ph[h] + (int64_t)(vsize*round(pitch*frame[k+1]*FMAXLEN/sr))) & PHMASK;
 amps[h] = frame[k];
}

static int perf_cudadsyn(CSOUND *csound, CUDADSYN *p){

  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t n, nsmps = CS_KSMPS;
  float *out_ = (float *) p->out_.auxp;
  MYFLT      *asig = p->asig;
  int count = p->count,  vsamps = p->vsamps;
  p->fp = (float *) (p->fsig->frame.auxp);

  if (UNLIKELY(offset)) memset(asig, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&asig[nsmps], '\0', early*sizeof(MYFLT));
   }

  for(n=offset; n < nsmps; n++){
    if(count == 0) {
      hipMemset(p->out, 0, sizeof(float)*vsamps);
      hipMemcpy(p->frame,p->fp,sizeof(float)*p->bins*2,hipMemcpyHostToDevice);
      sample<<<p->blocks,p->threads>>>(p->out,p->frame,
                                               *p->kfreq,
                                                p->ndx,
                                                p->previous,
                                                p->bins,
                                                vsamps,
                                                csound->GetSr(csound));
       if (hipDeviceSynchronize() != hipSuccess)
       csound->Message(csound,"Cuda error: Failed to synchronize\n");
       update<<<p->mblocks,p->mthreads>>>(p->frame,
                                           p->previous,
                                            p->ndx,
                                            *p->kfreq,
                                            vsamps,
                                            csound->GetSr(csound));
      hipMemcpy(out_,p->out,vsamps*sizeof(float),hipMemcpyDeviceToHost);
      count = vsamps;
    }
    asig[n] = (MYFLT) out_[vsamps - count];
    count--;
  }
  p->count = count;
  return OK;
}

static int destroy_cudadsyn(CSOUND *csound, void *pp){
  CUDADSYN *p = (CUDADSYN *) pp;
  hipFree(p->out);
  hipFree(p->ndx);
  hipFree(p->previous);
  hipFree(p->frame);
  return OK;
}


static OENTRY localops[] = {
  {"cudasynth", sizeof(CUDADSYN),0, 5, "a", "fkko", (SUBR) init_cudadsyn, NULL,
   (SUBR) perf_cudadsyn}
};

extern "C" {
  LINKAGE
}
