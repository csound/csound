#include "hip/hip_runtime.h"
// -*- c++ -*-
/* pvsops.cu
  experimental cuda opcodes

  (c) Victor Lazzarini, 2013

  based on M Puckette's pitch tracking algorithm.

  This file is part of Csound.

  The Csound Library is free software; you can redistribute it
  and/or modify it under the terms of the GNU Lesser General Public
  License as published by the Free Software Foundation; either
  version 2.1 of the License, or (at your option) any later version.

  Csound is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU Lesser General Public License for more details.

  You should have received a copy of the GNU Lesser General Public
  License along with Csound; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA
  02111-1307 USA
*/

#include <csdl.h>
#include <hipfft/hipfft.h>

#include <pstream.h>

/* kernel to convert from pvs to rectangular frame */
__global__ void frompvs(float* inframe, double* lastph,
                        double scal, double fac) {

  int k = threadIdx.x + blockIdx.x*blockDim.x + 1;
  int i = k << 1;
  float mag = inframe[i];
  double delta = (inframe[i+1] - k*scal)*fac;
  double phi = fmod(lastph[k-1] + delta, TWOPI);
  lastph[k-1] = phi;
  inframe[i] =  (float) (mag*cos(phi));
  inframe[i+1] = (float) (mag*sin(phi));
}

__global__ void winrotate(float* inframe2, float* inframe, float *win,
                          int N, int offset){
  int k = (threadIdx.x + blockIdx.x*blockDim.x);
  inframe2[k] = win[k]*inframe[(k+offset)%N];
}

typedef struct _pvsyn{
  OPDS h;
  MYFLT *asig;
  PVSDAT *fsig;
  float *inframe; /* N */
  float *inframe2;
  double *lastph;  /* N/2 */
  float *win;    /* N */
  int framecount;
  int curframe;
  AUXCH  frames;
  AUXCH  count;
  hipfftHandle plan;
  double scal, fac;
  int bblocks, nblocks;
  int bthreads, nthreads;
} PVSYN;

static int destroy_pvsyn(CSOUND *csound, void *pp);

static int pvsynset(CSOUND *csound, PVSYN *p){

  int N = p->fsig->N;

  if((N != 0) && !(N & (N - 1))) {
    int hsize = p->fsig->overlap;
    int size, numframes, i, blockspt;
    MYFLT sum = 0.0, bins = N/2;
    float *win;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    blockspt = deviceProp.maxThreadsPerBlock;
    csound->Message(csound, "CUDAsynth: using device %s (capability %d.%d)\n",
        deviceProp.name,deviceProp.major, deviceProp.minor);

    if(p->fsig->wintype != 1)
      return csound->InitError(csound,
                               "window type not implemented yet\n");
    numframes = N/hsize;
    size = N*sizeof(float)*numframes;
    if(p->frames.auxp == NULL ||
       p->frames.size < size)
      csound->AuxAlloc(csound, size, &p->frames);
    memset(p->frames.auxp, 0, size);

    size = sizeof(int)*numframes;
    if(p->count.auxp == NULL ||
       p->count.size < size)
      csound->AuxAlloc(csound, size, &p->count);
    *((int *)(p->count.auxp)) =  0;
    for(i=1; i < numframes; i++)
      ((int *)(p->count.auxp))[i] =
              (i + (1.f - (float)i/numframes))*N;

    size = (N+2)*sizeof(float);
    hipMalloc(&p->inframe, size);
    hipMemset(p->inframe, 0, size);
     size = (N+2)*sizeof(float);
    hipMalloc(&p->inframe2, size);
    size = (N+2)*sizeof(float);
    hipMalloc(&p->lastph, size);
    hipMemset(p->lastph, 0, size);
    size = N*sizeof(float);
    hipMalloc(&p->win, size);

    win = (float *) malloc(sizeof(float)*(N+1));
    for(i=0; i <= N; i++)
      win[i] = (float) (0.5 - 0.5*cos(i*TWOPI/N));

    for(i = 0; i < N; i++) sum += win[i];
    sum = FL(2.0) / sum;
    for(i = 0; i < N; i++) win[i] *= sum;
    sum = FL(0.0);
    for(i = 0; i <= N; i+=hsize)
               sum += win[i] * win[i];
    sum = (1.0/N)/(sum);
    for(i = 0; i < N; i++) win[i] *= 3*sum/sqrt(numframes);
    hipMemcpy(p->win,win,N*sizeof(float),
               hipMemcpyHostToDevice);
    free(win);

    p->framecount = 0;
    p->curframe = 0;

    p->fac = TWOPI*hsize/csound->GetSr(csound);
    p->scal =csound->GetSr(csound)/N;
    hipfftPlan1d(&p->plan, N, HIPFFT_C2R, 1);
    cufftSetCompatibilityMode(p->plan, CUFFT_COMPATIBILITY_NATIVE);
    csound->RegisterDeinitCallback(csound, p, destroy_pvsyn);

    p->bblocks = bins > blockspt? bins/blockspt : 1;
    p->nblocks = N > blockspt ? N/blockspt : 1;
    p->bthreads = bins/p->bblocks;
    p->nthreads = N/p->nblocks;
    if(csound->GetDebug(csound))
      csound->Message(csound, "%d (%d each), %d (%d each)\n",
                    p->nblocks, p->nthreads, p->bblocks, p->bthreads);
    return OK;
  }
  return csound->InitError(csound, "fftsize not power-of-two \n");

}

static int pvsynperf(CSOUND *csound, PVSYN *p){

  int N = p->fsig->N, i;
  int hsize = p->fsig->overlap;
  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t n, nsmps = CS_KSMPS;
  MYFLT *asig = p->asig;
  float *frames = (float *) p->frames.auxp;
  int framecount = p->framecount;
  int numframes = N/hsize;
  int *count = (int *) p->count.auxp;

  if (UNLIKELY(offset)) memset(asig, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&asig[nsmps], '\0', early*sizeof(MYFLT));
  }

  for(n=offset; n < nsmps; n++) {

    if(framecount == 0) {
      int curframe = p->curframe;
      /* start offset for current frame */
      int start = N*curframe;
      float *cur = &(frames[start]);
      float *win = (float *) p->win;
      float *inframe = p->inframe;
            float *inframe2 = p->inframe2;
      float *fsig = (float *) p->fsig->frame.auxp;
      /* copy fsig data to device */
      fsig[N+1] = fsig[1] = 0.f;
      hipMemcpy(inframe,fsig,(N+2)*sizeof(float),
                 hipMemcpyHostToDevice);
      /* perf pvs to rect conversion */
      frompvs<<<p->bblocks,p->bthreads-1>>>(inframe,p->lastph,p->scal,p->fac);
      /* execute inverse real FFT */
      if(hipfftExecC2R(p->plan,(hipfftComplex*)inframe,inframe)
         != HIPFFT_SUCCESS) csound->Message(csound, "cuda fft error\n");
      if (hipDeviceSynchronize() != hipSuccess)
        csound->Message(csound,"Cuda error: Failed to synchronize\n");
      /* window and rotate data on device */
      winrotate<<<p->nblocks,p->nthreads>>>(inframe2,inframe,win,N,hsize*curframe);
      /* copy data to current out frame */
      hipMemcpy(cur,inframe2,N*sizeof(float),hipMemcpyDeviceToHost);
      /* reset counter for this frame to the start */
      count[curframe] = start;
      /* move current to next frame circularly */
      p->curframe = ++(curframe) == numframes ? 0 : curframe;
      framecount = hsize;
    }
    asig[n] = FL(0.0);
    for(i=0; i < numframes; i++){
      /* overlap-add */
      asig[n] += frames[count[i]];
      count[i]++;
    }
    framecount--;
  }
  p->framecount = framecount;
  return OK;
}

static int destroy_pvsyn(CSOUND *csound, void *pp){
  PVSYN *p = (PVSYN *) pp;
  hipfftDestroy(p->plan);
  hipFree(p->inframe);
  hipFree(p->inframe2);
  hipFree(p->lastph);
  hipFree(p->win);
  return OK;
}

__device__ double modTwoPi(double x)
{
  x = fmod(x,TWOPI);
  return x <= -PI ? x + TWOPI :
    (x > PI ? x - TWOPI : x);
}

/* kernel to convert from rectangular to pvs frame */
__global__ void topvs(float* aframe, double* oldph,
                      double scal, double fac) {
  int k = threadIdx.x + blockIdx.x*blockDim.x + 1;
  int i = k << 1;

  float re = aframe[i], im = aframe[i+1];
  float mag = sqrtf(re*re + im*im);
  double phi = atan2f(im,re);
  double delta = phi - oldph[k-1];
  oldph[k-1] = phi;
  aframe[i] =  mag;
  aframe[i+1] = (float) ((modTwoPi(delta) + k*scal)*fac);
}

__global__ void rotatewin(float* aframe2, float *aframe, float *win,
                          int N, int offset){
  int k = threadIdx.x + blockIdx.x*blockDim.x;
  aframe2[(k+offset)%N] = win[k]*aframe[k];
}

typedef struct _pvan {
  OPDS  h;
  PVSDAT *fsig;
  MYFLT *asig,*fftsize,*hsize,*winsize,*wintype;

  float *aframe; /* N */
  float *aframe2;
  double *oldph;  /* N/2 */
  float *win;    /* N */

  int framecount;
  int curframe;
  AUXCH  frames;
  AUXCH  count;
  hipfftHandle plan;
  double scal, fac;
  int bblocks, nblocks;
  int bthreads, nthreads;
} PVAN;

static int destroy_pvanal(CSOUND *csound, void *pp);

static int pvanalset(CSOUND *csound, PVAN *p){

  int N = *p->fftsize;
  if((N != 0) && !(N & (N - 1))) {
    int size, numframes, i, bins = N/2;
    int hsize = *p->hsize, blockspt;
    float *win;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    blockspt = deviceProp.maxThreadsPerBlock;
    csound->Message(csound, "CUDAnal: using device %s (capability %d.%d)\n",
        deviceProp.name,deviceProp.major, deviceProp.minor);

    p->fsig->N = N;
    p->fsig->overlap = hsize;
    /* ignore winsize & wintype */
    p->fsig->winsize = N;
    p->fsig->wintype = 1;
    p->fsig->framecount = 0;

    numframes = N/hsize;
    size = N*sizeof(float)*numframes;
    if(p->frames.auxp == NULL ||
       p->frames.size < size)
      csound->AuxAlloc(csound, size, &p->frames);
    memset(p->frames.auxp, 0, size);

    size = (N+2)*sizeof(float);
    if(p->fsig->frame.auxp == NULL ||
       p->fsig->frame.size < size)
      csound->AuxAlloc(csound, size, &p->fsig->frame);
    memset(p->fsig->frame.auxp, 0, size);

    size = sizeof(int)*numframes;
    if(p->count.auxp == NULL ||
       p->count.size < size)
      csound->AuxAlloc(csound, size, &p->count);
    *((int *)(p->count.auxp)) =  0;
    for(i=1; i < numframes; i++)
      ((int *)(p->count.auxp))[i] =
              (i + (float)i/numframes)*N;

    size = (N+2)*sizeof(float);
    hipMalloc(&p->aframe, size);
    size = (N+2)*sizeof(float);
    hipMalloc(&p->aframe2, size);
    size = (N/2-1)*sizeof(double);
    hipMalloc(&p->oldph, size);
    hipMemset(p->oldph, 0, size);
    size = N*sizeof(float);
    hipMalloc(&p->win, size);

    win = (float *) malloc(sizeof(float)*N);
    for(i=0; i < N; i++)
      win[i] = (float) (0.5 - 0.5*cos(i*TWOPI/N));
    float sum = 0.0;
   for(i = 0; i < N; i++) sum += win[i];
    sum = FL(2.0) / sum;
   for(i = 0; i < N; i++) win[i] *= sum;

    hipMemcpy(p->win,win,N*sizeof(float),
               hipMemcpyHostToDevice);
    free(win);

    p->framecount = 1;
    p->curframe = numframes-1;
    p->fac = csound->GetSr(csound)/(TWOPI*hsize);
    p->scal = (TWOPI*hsize)/N;
    hipfftPlan1d(&p->plan, N, HIPFFT_R2C, 1);
    cufftSetCompatibilityMode(p->plan, CUFFT_COMPATIBILITY_NATIVE);
    csound->RegisterDeinitCallback(csound, p, destroy_pvanal);

    p->bblocks = bins > blockspt? bins/blockspt : 1;
    p->nblocks = N > blockspt ? N/blockspt : 1;
    p->bthreads = bins/p->bblocks;
    p->nthreads = N/p->nblocks;
  if(csound->GetDebug(csound))
    csound->Message(csound, "%d (%d each), %d (%d each)\n",
                    p->nblocks, p->nthreads, p->bblocks, p->bthreads);
    return OK;
  }
  return csound->InitError(csound, "fftsize not power-of-two \n");
}

static int pvanalperf(CSOUND *csound, PVAN *p){

  int N = p->fsig->N, i;
  int hsize = p->fsig->overlap;
  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t n, nsmps = CS_KSMPS;
  MYFLT *asig = p->asig;
  float *frames = (float *) p->frames.auxp;
  int framecount = p->framecount;
  int numframes = N/hsize;
  int *count = (int *) p->count.auxp;

  if (UNLIKELY(offset)) memset(asig, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&asig[nsmps], '\0', early*sizeof(MYFLT));
  }

  for(n=offset; n < nsmps; n++) {
   for(i=0; i < numframes; i++){
      frames[count[i]] = asig[n];
      count[i]++;
    }
    framecount++;

    if(framecount == hsize) {
      int curframe = p->curframe;
      /* start offset for current frame */
      int start = N*curframe;
      float *cur = &(frames[start]);
      float *win = (float *) p->win;
      float *aframe = p->aframe;
      float *aframe2 = p->aframe2;
      float *fsig = (float *) p->fsig->frame.auxp;
      /* copy fsig data to device */
      hipMemcpy(aframe,cur,N*sizeof(float),
             hipMemcpyHostToDevice);
      /* window and rotate data on device */
      rotatewin<<<p->nblocks,p->nthreads>>>(aframe2,aframe,win,N,
                                            hsize*(numframes-curframe));
       /* execute inverse real FFT */
      if(hipfftExecR2C(p->plan,aframe2,(hipfftComplex*)aframe2)
      != HIPFFT_SUCCESS) csound->Message(csound, "cuda fft error\n");
       if (hipDeviceSynchronize() != hipSuccess)
         csound->Message(csound,"Cuda error: Failed to synchronize\n");
       /* perf rect to pvs conversion */
       topvs<<<p->bblocks,p->bthreads-1>>>(aframe2,p->oldph,p->scal,p->fac);
       /* copy data to current out frame */
      hipMemcpy(fsig,aframe2,(N+2)*sizeof(float),hipMemcpyDeviceToHost);
      /* reset counter for this frame to the start */
      fsig[N+1] = fsig[1] = 0.f;
      count[curframe] = start;
      /* move current to next frame circularly */
      p->curframe = --(curframe) < 0 ? numframes-1 : curframe;
      framecount = 0;
      p->fsig->framecount++;
    }
  }
  p->framecount = framecount;
  return OK;
}

static int destroy_pvanal(CSOUND *csound, void *pp){
  PVAN *p = (PVAN *) pp;
  hipfftDestroy(p->plan);
  hipFree(p->aframe);
  hipFree(p->aframe2);
  hipFree(p->oldph);
  hipFree(p->win);
  return OK;
}

static OENTRY localops[] = {
  {"cudasynth", sizeof(PVSYN),0, 5, "a", "f", (SUBR) pvsynset, NULL,
   (SUBR) pvsynperf},
   {"cudanal", sizeof(PVAN),0, 5, "f", "aiiii", (SUBR) pvanalset, NULL,
   (SUBR) pvanalperf}
};

extern "C" {
  LINKAGE
}
