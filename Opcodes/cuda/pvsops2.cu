#include "hip/hip_runtime.h"
// -*- c++ -*-
/* pvsops.cu
  experimental cuda opcodes

  (c) Victor Lazzarini, 2013

  based on M Puckette's pitch tracking algorithm.

  This file is part of Csound.

  The Csound Library is free software; you can redistribute it
  and/or modify it under the terms of the GNU Lesser General Public
  License as published by the Free Software Foundation; either
  version 2.1 of the License, or (at your option) any later version.

  Csound is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU Lesser General Public License for more details.

  You should have received a copy of the GNU Lesser General Public
  License along with Csound; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA
  02111-1307 USA
*/

#include <csdl.h>
#include <hipfft/hipfft.h>

#include <pstream.h>

void AuxCudaAlloc(int size, AUXCH *p){
  float *mem;
  hipMalloc(&mem, size);
  hipMemset(mem, 0, size);  
  p->auxp = mem;
  p->size = size;
}


/* kernel to convert from pvs to rectangular frame */
__global__ void frompvs(float* inframe, float* fsig, double* lastph,
                        double scal, double fac) {

  int k = threadIdx.x + blockIdx.x*blockDim.x;
  int i = k << 1;
  float mag = fsig[i];
  double delta = (fsig[i+1] - k*scal)*fac;
  double phi = fmod(lastph[k] + delta, TWOPI);
  lastph[k] = phi;
  inframe[i] =  (float) (mag*cos(phi));
  inframe[i+1] = (float) (mag*sin(phi));
}

__global__ void winrotate(float* inframe2, float* inframe, float *win,
                          int N, int offset){
  int k = (threadIdx.x + blockIdx.x*blockDim.x);
  inframe2[k] = win[k]*inframe[(k+offset)%N];
}

typedef struct _pvsyn{
  OPDS h;
  MYFLT *asig;
  PVSDAT *fsig;
  float *inframe; /* N */
  float *inframe2;
  double *lastph;  /* N/2 */
  float *win;    /* N */
  int framecount;
  int curframe;
  AUXCH  frames;
  AUXCH  count;
  hipfftHandle plan;
  double scal, fac;
  int bblocks, nblocks;
  int bthreads, nthreads;
} PVSYN;

static int destroy_pvsyn(CSOUND *csound, void *pp);

static int pvsynset(CSOUND *csound, PVSYN *p){

  int N = p->fsig->N;

  if((N != 0) && !(N & (N - 1))) {
    int hsize = p->fsig->overlap;
    int size, numframes, i, blockspt;
    MYFLT sum = 0.0, bins = N/2;
    float *win;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    blockspt = deviceProp.maxThreadsPerBlock;
    csound->Message(csound, "CUDAsynth: using device %s (capability %d.%d)\n",
        deviceProp.name,deviceProp.major, deviceProp.minor);

    if(p->fsig->wintype != 1)
      return csound->InitError(csound,
                               "window type not implemented yet\n");
    numframes = N/hsize;
    size = N*sizeof(float)*numframes;
    if(p->frames.auxp == NULL ||
       p->frames.size < size)
      csound->AuxAlloc(csound, size, &p->frames);
    memset(p->frames.auxp, 0, size);

    size = sizeof(int)*numframes;
    if(p->count.auxp == NULL ||
       p->count.size < size)
      csound->AuxAlloc(csound, size, &p->count);
    *((int *)(p->count.auxp)) =  0;
    for(i=1; i < numframes; i++)
      ((int *)(p->count.auxp))[i] =
              (i + (1.f - (float)i/numframes))*N;

    size = (N+2)*sizeof(float);
    hipMalloc(&p->inframe, size);
    hipMemset(p->inframe, 0, size);
    size = (N+2)*sizeof(float);
    hipMalloc(&p->inframe2, size);
    size = (N/2)*sizeof(double);
    hipMalloc(&p->lastph, size);
    hipMemset(p->lastph, 0, size);
    size = N*sizeof(float);
    hipMalloc(&p->win, size);

    win = (float *) malloc(sizeof(float)*(N+1));
    for(i=0; i <= N; i++)
      win[i] = (float) (0.5 - 0.5*cos(i*TWOPI/N));

    for(i = 0; i < N; i++) sum += win[i];
    sum = FL(2.0) / sum;
    for(i = 0; i < N; i++) win[i] *= sum;
    sum = FL(0.0);
    for(i = 0; i <= N; i+=hsize)
               sum += win[i] * win[i];
    sum = (1.0/N)/(sum);
    for(i = 0; i < N; i++) win[i] *= 3*sum/sqrt(numframes);
    hipMemcpy(p->win,win,N*sizeof(float),
               hipMemcpyHostToDevice);
    free(win);

    p->framecount = 0;
    p->curframe = 0;

    p->fac = TWOPI*hsize/csound->GetSr(csound);
    p->scal =csound->GetSr(csound)/N;
    hipfftPlan1d(&p->plan, N, HIPFFT_C2R, 1);
    cufftSetCompatibilityMode(p->plan, CUFFT_COMPATIBILITY_NATIVE);
    csound->RegisterDeinitCallback(csound, p, destroy_pvsyn);

    p->bblocks = bins > blockspt? bins/blockspt : 1;
    p->nblocks = N > blockspt ? N/blockspt : 1;
    p->bthreads = bins/p->bblocks;
    p->nthreads = N/p->nblocks;
    if(csound->GetDebug(csound))
      csound->Message(csound, "%d (%d each), %d (%d each)\n",
                    p->nblocks, p->nthreads, p->bblocks, p->bthreads);
    return OK;
  }
  return csound->InitError(csound, "fftsize not power-of-two \n");

}

static int pvsynperf(CSOUND *csound, PVSYN *p){

  int N = p->fsig->N, i;
  int hsize = p->fsig->overlap;
  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t n, nsmps = CS_KSMPS;
  MYFLT *asig = p->asig;
  float *frames = (float *) p->frames.auxp;
  int framecount = p->framecount;
  int numframes = N/hsize;
  int *count = (int *) p->count.auxp;

  if (UNLIKELY(offset)) memset(asig, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&asig[nsmps], '\0', early*sizeof(MYFLT));
  }

  for(n=offset; n < nsmps; n++) {

    if(framecount == 0) {
      int curframe = p->curframe;
      /* start offset for current frame */
      int start = N*curframe;
      float *cur = &(frames[start]);
      float *win = (float *) p->win;
      float *inframe = p->inframe;
      float *inframe2 = p->inframe2;
      float *fsig = (float *) p->fsig->frame.auxp;
      /* perf pvs to rect conversion */
      frompvs<<<p->bblocks,p->bthreads>>>(inframe,fsig,p->lastph,p->scal,p->fac);
      /* execute inverse real FFT */
      if(hipfftExecC2R(p->plan,(hipfftComplex*)inframe,inframe)
         != HIPFFT_SUCCESS) csound->Message(csound, "cuda fft error\n");
      if (hipDeviceSynchronize() != hipSuccess)
        csound->Message(csound,"Cuda error: Failed to synchronize\n");
      /* window and rotate data on device */
      winrotate<<<p->nblocks,p->nthreads>>>(inframe2,inframe,win,N,hsize*curframe);
      /* copy data to current out frame */
      hipMemcpy(cur,inframe2,N*sizeof(float),hipMemcpyDeviceToHost);
      /* reset counter for this frame to the start */
      count[curframe] = start;
      /* move current to next frame circularly */
      p->curframe = ++(curframe) == numframes ? 0 : curframe;
      framecount = hsize;
    }
    asig[n] = FL(0.0);
    for(i=0; i < numframes; i++){
      /* overlap-add */
      asig[n] += frames[count[i]];
      count[i]++;
    }
    framecount--;
  }
  p->framecount = framecount;
  return OK;
}

static int destroy_pvsyn(CSOUND *csound, void *pp){
  PVSYN *p = (PVSYN *) pp;
  hipfftDestroy(p->plan);
  hipFree(p->inframe);
  hipFree(p->inframe2);
  hipFree(p->lastph);
  hipFree(p->win);
  return OK;
}

__device__ double modTwoPi(double x)
{
  x = fmod(x,TWOPI);
  return x <= -PI ? x + TWOPI :
    (x > PI ? x - TWOPI : x);
}

/* kernel to convert from rectangular to pvs frame */
__global__ void topvs(float *fsig, float *aframe, double* oldph,
                      double scal, double fac) {
  int k = threadIdx.x + blockIdx.x*blockDim.x;
  int i = k << 1;
  float re = aframe[i], im = aframe[i+1];
  float mag = sqrtf(re*re + im*im);
  double phi = atan2f(im,re);
  double delta = phi - oldph[k];
  oldph[k] = phi;
  fsig[i] =  mag;
  fsig[i+1] = (float) ((modTwoPi(delta) + k*scal)*fac);
}

__global__ void rotatewin(float* aframe2, float *aframe, float *win,
                          int N, int offset){
  int k = threadIdx.x + blockIdx.x*blockDim.x;
  aframe2[(k+offset)%N] = win[k]*aframe[k];
}

typedef struct _pvan {
  OPDS  h;
  PVSDAT *fsig;
  MYFLT *asig,*fftsize,*hsize,*winsize,*wintype;

  float *aframe; /* N */
  float *aframe2;
  double *oldph;  /* N/2 */
  float *win;    /* N */

  int framecount;
  int curframe;
  AUXCH  frames;
  AUXCH  count;
  hipfftHandle plan;
  double scal, fac;
  int bblocks, nblocks;
  int bthreads, nthreads;
} PVAN;

static int destroy_pvanal(CSOUND *csound, void *pp);

static int pvanalset(CSOUND *csound, PVAN *p){

  int N = *p->fftsize;
  if((N != 0) && !(N & (N - 1))) {
    int size, numframes, i, bins = N/2;
    int hsize = *p->hsize, blockspt;
    float *win;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    blockspt = deviceProp.maxThreadsPerBlock;
    csound->Message(csound, "CUDAnal: using device %s (capability %d.%d)\n",
        deviceProp.name,deviceProp.major, deviceProp.minor);

    p->fsig->N = N;
    p->fsig->overlap = hsize;
    p->fsig->format = -1;
    /* ignore winsize & wintype */
    p->fsig->winsize = N;
    p->fsig->wintype = 1;
    p->fsig->framecount = 0;

    numframes = N/hsize;
    size = N*sizeof(float)*numframes;
    if(p->frames.auxp == NULL ||
       p->frames.size < size)
      csound->AuxAlloc(csound, size, &p->frames);
    memset(p->frames.auxp, 0, size);

    size = (N+2)*sizeof(float);
    if(p->fsig->frame.auxp == NULL ||
       p->fsig->frame.size < size)
       AuxCudaAlloc(size, &p->fsig->frame);
    
    size = sizeof(int)*numframes;
    if(p->count.auxp == NULL ||
       p->count.size < size)
      csound->AuxAlloc(csound, size, &p->count);
    *((int *)(p->count.auxp)) =  0;
    for(i=1; i < numframes; i++)
      ((int *)(p->count.auxp))[i] =
              (i + (float)i/numframes)*N;

    size = (N+2)*sizeof(float);
    hipMalloc(&p->aframe, size);
    size = (N+2)*sizeof(float);
    hipMalloc(&p->aframe2, size);
    size = (N/2)*sizeof(double);
    hipMalloc(&p->oldph, size);
    hipMemset(p->oldph, 0, size);
    size = N*sizeof(float);
    hipMalloc(&p->win, size);


    win = (float *) malloc(sizeof(float)*N);
    for(i=0; i < N; i++)
      win[i] = (float) (0.5 - 0.5*cos(i*TWOPI/N));
    float sum = 0.0;
   for(i = 0; i < N; i++) sum += win[i];
    sum = FL(2.0) / sum;
   for(i = 0; i < N; i++) win[i] *= sum;

    hipMemcpy(p->win,win,N*sizeof(float),
               hipMemcpyHostToDevice);
    free(win);

    p->framecount = 1;
    p->curframe = numframes-1;
    p->fac = csound->GetSr(csound)/(TWOPI*hsize);
    p->scal = (TWOPI*hsize)/N;
    hipfftPlan1d(&p->plan, N, HIPFFT_R2C, 1);
    cufftSetCompatibilityMode(p->plan, CUFFT_COMPATIBILITY_NATIVE);
    csound->RegisterDeinitCallback(csound, p, destroy_pvanal);

    p->bblocks = bins > blockspt? bins/blockspt : 1;
    p->nblocks = N > blockspt ? N/blockspt : 1;
    p->bthreads = bins/p->bblocks;
    p->nthreads = N/p->nblocks;
   if(csound->GetDebug(csound))
    csound->Message(csound, "%d (%d each), %d (%d each)\n",
                    p->nblocks, p->nthreads, p->bblocks, p->bthreads);
    return OK;
  }
  return csound->InitError(csound, "fftsize not power-of-two \n");
}

static int pvanalperf(CSOUND *csound, PVAN *p){

  int N = p->fsig->N, i;
  int hsize = p->fsig->overlap;
  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t n, nsmps = CS_KSMPS;
  MYFLT *asig = p->asig;
  float *frames = (float *) p->frames.auxp;
  int framecount = p->framecount;
  int numframes = N/hsize;
  int *count = (int *) p->count.auxp;

  if (UNLIKELY(offset)) memset(asig, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&asig[nsmps], '\0', early*sizeof(MYFLT));
  }

  for(n=offset; n < nsmps; n++) {
   for(i=0; i < numframes; i++){
      frames[count[i]] = asig[n];
      count[i]++;
    }
    framecount++;

    if(framecount == hsize) {
      int curframe = p->curframe;
      /* start offset for current frame */
      int start = N*curframe;
      float *cur = &(frames[start]);
      float *win = (float *) p->win;
      float *aframe = p->aframe;
      float *aframe2 = p->aframe2;
      float *fsig = (float *) p->fsig->frame.auxp;
      hipMemcpy(aframe,cur,N*sizeof(float),
             hipMemcpyHostToDevice);
      /* window and rotate data on device */
      rotatewin<<<p->nblocks,p->nthreads>>>(aframe2,aframe,win,N,
                                            hsize*(numframes-curframe));
       /* execute inverse real FFT */
      if(hipfftExecR2C(p->plan,aframe2,(hipfftComplex*)aframe2)
      != HIPFFT_SUCCESS) csound->Message(csound, "cuda fft error\n");
       if (hipDeviceSynchronize() != hipSuccess)
         csound->Message(csound,"Cuda error: Failed to synchronize\n");
       /* perf rect to pvs conversion */
       topvs<<<p->bblocks,p->bthreads>>>(fsig,aframe2,p->oldph,p->scal,p->fac);
      /* reset counter for this frame to the start */
      count[curframe] = start;
      /* move current to next frame circularly */
      p->curframe = --(curframe) < 0 ? numframes-1 : curframe;
      framecount = 0;
      p->fsig->framecount++;
    }
  }
  p->framecount = framecount;
  return OK;
}

static int destroy_pvanal(CSOUND *csound, void *pp){
  PVAN *p = (PVAN *) pp;
  hipfftDestroy(p->plan);
  hipFree(p->aframe);
  hipFree(p->aframe2);
  hipFree(p->oldph);
  hipFree(p->fsig->frame.auxp);
  p->fsig->frame.size = 0;
  hipFree(p->win);
  return OK;
}

typedef struct _cudapvsgain2 {
  OPDS    h;
  PVSDAT  *fout;
  PVSDAT  *fa;
  MYFLT   *kgain;
  int gridSize;   // number of blocks in the grid (1D)
  int blockSize;   // number of threads in one block (1D)
  uint32  lastframe;
} CUDAPVSGAIN2;

// kernel for scaling PV amplitudes
__global__ void applygain(float* output, float* input, MYFLT g, int length) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int j = i<<1;

  if(j < length){
    output[j] = (float) input[j] * g;
    output[j+1] = input[j+1];
  }
} 

static int free_device(CSOUND* csound, void* pp){
  CUDAPVSGAIN2* p = (CUDAPVSGAIN2*) pp;
  hipFree(p->fout->frame.auxp);
  return OK;
} 

static int cudapvsgain2set(CSOUND *csound, CUDAPVSGAIN2 *p){

  int32 N = p->fa->N;
  int size = (N+2) * sizeof(float);
  int maxBlockDim;
  int SMcount;
  int totNumThreads = (N+2)/2;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp,0);
  maxBlockDim = deviceProp.maxThreadsPerBlock;
  SMcount = deviceProp.multiProcessorCount;
  csound->Message(csound, "cudapvsgain2 running on device %s (capability %d.%d)\n", deviceProp.name,
     deviceProp.major, deviceProp.minor);
    
  p->fout->sliding = 0;
    
  if (p->fout->frame.auxp == NULL || p->fout->frame.size < size)
    AuxCudaAlloc(size, &p->fout->frame);
    
  p->blockSize = (((totNumThreads/SMcount)/32)+1)*32;
  if (p->blockSize > maxBlockDim) p->blockSize = maxBlockDim;
  p->gridSize = totNumThreads / p->blockSize + 1;  
  p->fout->N = N;
  p->fout->overlap = p->fa->overlap;
  p->fout->winsize = p->fa->winsize;
  p->fout->wintype = p->fa->wintype;
  p->fout->format = p->fa->format;
  p->fout->framecount = 1;
  p->lastframe = 0;
  
  csound->RegisterDeinitCallback(csound, p, free_device);
  
  return OK;
}

static int cudapvsgain2(CSOUND *csound, CUDAPVSGAIN2 *p)
{
  int32   framelength = p->fa->N + 2;
  MYFLT gain = *p->kgain;
  float* fo = (float*) p->fout->frame.auxp;
  float* fi = (float*) p->fa->frame.auxp;

  if (p->lastframe < p->fa->framecount) {
     if (hipDeviceSynchronize() != hipSuccess)
         csound->Message(csound,"Cuda error: Failed to synchronize\n");
    applygain<<<p->gridSize,p->blockSize>>>(fo, fi, gain, framelength); 
    p->fout->framecount = p->fa->framecount;
    p->lastframe = p->fout->framecount;
  }

  return OK;
}


static OENTRY localops[] = {
  {"cudasynth2", sizeof(PVSYN),0, 5, "a", "f", (SUBR) pvsynset, NULL,
   (SUBR) pvsynperf},
   {"cudanal2", sizeof(PVAN),0, 5, "f", "aiiii", (SUBR) pvanalset, NULL,
    (SUBR) pvanalperf},
  {"cudapvsgain2", sizeof(CUDAPVSGAIN2), 0, 3, "f", "fk",
                               (SUBR) cudapvsgain2set, (SUBR) cudapvsgain2, NULL}
};

extern "C" {
  LINKAGE
}
